#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <vector>
#include <c10/cuda/CUDAStream.h>

#include "../include/merged_spmv.cuh"

// Caching allocator for device memory
hipcub::CachingDeviceAllocator g_allocator(true);

template <typename ValueT, typename OffsetT>
torch::Tensor launch_flex_spmv_cuda(
    torch::Tensor spm_k,
    torch::Tensor spm_l,
    torch::Tensor row_offsets,
    torch::Tensor col_indices_i,
    torch::Tensor col_indices_j,
    torch::Tensor vector_x,
    torch::Tensor vector_y,
    int num_rows,
    int num_cols,
    int num_nonzeros) {
    
    // Setup FlexParams struct with PyTorch tensor data
    FlexParams<ValueT, OffsetT> params;
    params.spm_k_ptr = spm_k.data_ptr<ValueT>();
    // params.d_spm_nnz = spm_l.data_ptr<ValueT>(); // use it later
    params.row_end_offsets_ptr = row_offsets.data_ptr<OffsetT>();
    params.selector_i_ptr = col_indices_i.data_ptr<OffsetT>();
    params.selector_j_ptr = col_indices_j.data_ptr<OffsetT>();
    params.vector_x_ptr = vector_x.data_ptr<ValueT>();
    params.d_vector_y = vector_y.data_ptr<ValueT>();
    params.num_rows = num_rows;
    params.num_cols = num_cols;
    params.num_nonzeros = num_nonzeros;
    
    // Get current CUDA stream from PyTorch
    hipStream_t stream = c10::cuda::getCurrentCUDAStream();
    
    // Allocate temporary storage
    size_t temp_storage_bytes = 0;
    void *d_temp_storage = nullptr;
    
    // Get amount of temporary storage needed
    hipError_t error = merged::merged_spmv_launch<ValueT, OffsetT>(
        params, d_temp_storage, temp_storage_bytes, false, stream);
    
    if (error != hipSuccess) {
        throw std::runtime_error("Error in merged_spmv_launch: " + 
                                std::string(hipGetErrorString(error)));
    }
    
    // Allocate temporary storage
    error = g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes);
    if (error != hipSuccess) {
        throw std::runtime_error("Error allocating temporary storage: " + 
                                std::string(hipGetErrorString(error)));
    }
    
    // Launch the SpMV kernel
    error = merged::merged_spmv_launch<ValueT, OffsetT>(
        params, d_temp_storage, temp_storage_bytes, false, stream);
    
    if (error != hipSuccess) {
        g_allocator.DeviceFree(d_temp_storage);
        throw std::runtime_error("Error in merged_spmv_launch: " + 
                                std::string(hipGetErrorString(error)));
    }
    
    // Free temporary storage
    g_allocator.DeviceFree(d_temp_storage);
    
    return vector_y;
}

// Explicit instantiation for float and double types
template torch::Tensor launch_flex_spmv_cuda<float, int>(
    torch::Tensor spm_k,
    torch::Tensor spm_l,
    torch::Tensor row_offsets,
    torch::Tensor col_indices_i,
    torch::Tensor col_indices_j,
    torch::Tensor vector_x,
    torch::Tensor vector_y,
    int num_rows,
    int num_cols,
    int num_nonzeros);

template torch::Tensor launch_flex_spmv_cuda<double, int>(
    torch::Tensor spm_k,
    torch::Tensor spm_l,
    torch::Tensor row_offsets,
    torch::Tensor col_indices_i,
    torch::Tensor col_indices_j,
    torch::Tensor vector_x,
    torch::Tensor vector_y,
    int num_rows,
    int num_cols,
    int num_nonzeros); 